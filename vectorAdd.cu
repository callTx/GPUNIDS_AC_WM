#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

#include <vector>
#include <iostream>
#include <string>
#include <cstring>
#include <queue>
#include <fstream>
#include <algorithm>
#include <math.h>
#include <stdio.h>

int TmhTodosPadroes();
int TmhMenorPadrao();
double BlcStr();
void vecPermu1_1();
void PrecarregaTabelaHash(std::string strTemp, int hash, char* dataTemp);

bool hashEsgotado(int hash);

int tbHDataSize = 0;
std::vector<std::string> padroes;
std::vector<int> arrHashEsgotados;
int* tbHContPadHash;
int* tbHPosData;
int* tbHSizePadHash;
int* tbHIdxSizePadHash;
int countSizePadHash = 0;


int B;
int M;
int c = 6;// 128
int m;
int nP;

char* tbHData;
int conttbHData = 0;
int indexDebug;
int THREADS_PER_BLOCk;
int NUMBER_BLOCKS;

char* d_tbHData = NULL;
int* d_tbHPosData = NULL;
int* d_tbHContPadHash = NULL;
int* d_tbHSizePadHash = NULL;
int* d_tbHIdxSizePadHash = NULL;

using namespace std;
/*
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}
*/
/**
 * Host main routine
 */

/*
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}
*/

__global__ void
wmkernel(char* dataTemp, int* tbHContPadHash, int* tbHPosData, int* tbHSizePadHash, int* tbHIdxSizePadHash, char* tbHData, int N, double B, int c, int M, int* posicao) {

	int index = blockDim.x * blockIdx.x + threadIdx.x;




	if (index < N - B + 1) {
		//posicao[index] = index;

		int hash = 0;
		for (int i = index; i < index + B; i++) {
			hash = hash + dataTemp[i] * (c ^ (i - index));
			hash = hash % M;
		}
		//posicao[index] = hash;

		int tamArrPadrao = 0;
		for (unsigned t = 0; t < tbHContPadHash[hash]; t++) {
			int match = 1;

			int posIniPadrao = tbHPosData[hash] - 1;
			tamArrPadrao = tamArrPadrao + tbHSizePadHash[tbHIdxSizePadHash[hash] + t];
			int tamPadrao = tbHSizePadHash[tbHIdxSizePadHash[hash] + t];
			int _final = (index + B) - 1;
			int _final2 = tamPadrao;
			for (int ch = tamArrPadrao; ch >= 0; --ch) {
				if (_final2 - 1 >= 0) {

					if (tbHData[posIniPadrao + ch - 1] == dataTemp[_final]) {
						_final = _final - 1;
						_final2 = _final2 - 1;

					}
					else {
						match = 0;
						break;
					}
				}
				else {

					break;
				}
			}
			if (match) {
				posicao[index] = _final + 1;


			}
		}


	}


}

void execGPU() {
	
	std::cout << std::endl;
	std::cout << "Inicio exec" << std::endl;

	hipError_t err = hipSuccess;

	//========== aloca host, aloca device dataTemp ==========
	char* dataTemp;
	dataTemp = "Accept|3A|gk3Connectionkkjwf/;.bCer.93senduuname";

	int N = strlen(dataTemp);
	size_t sizeChar = N * sizeof(char);

	char* d_dataTemp = NULL;
	err = hipMalloc((void**)& d_dataTemp, sizeChar);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_dataTemp (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//==========fim aloca host, aloca device dataTemp ==========

	//============ copia device dataTemp ===============================
	err = hipMemcpy(d_dataTemp, dataTemp, sizeChar, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector dataTemp from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//============fim copia device dataTemp ===============================

	//=========== aloca host,aloca device posicao===============
	size_t size = N * sizeof(int);
	int* posicao;
	int* d_posicao = NULL;

	posicao = (int*)calloc(N, size);
	hipMalloc((void**)& d_posicao, size);
	//===========fim: aloca host,aloca device posicao===============

	//======== verifica se N e multiplo de THREADS_PER_BLOCk ======================
	NUMBER_BLOCKS = N / THREADS_PER_BLOCk;
	cout << "\n";
	cout << "NUMBER_BLOCKS: " << NUMBER_BLOCKS << "\n";

	int X_;
	int Y_;
	if (N % THREADS_PER_BLOCk == 0) {
		cout << "N/ THREADS_PER_BLOCk: " << N / THREADS_PER_BLOCk << " e multiplo";
		X_ = N / THREADS_PER_BLOCk;
		Y_ = THREADS_PER_BLOCk;
	}
	else {
		cout << "N/ THREADS_PER_BLOCk: " << N / THREADS_PER_BLOCk << " noa e multiplo";
		X_ = (NUMBER_BLOCKS + THREADS_PER_BLOCk - 1) / THREADS_PER_BLOCk;
		Y_ = THREADS_PER_BLOCk;
	}
	//========fim verifica se N e multiplo de THREADS_PER_BLOCk ======================


	//=======executa o kernel=====================================================
	wmkernel<<< X_, Y_ >>>(d_dataTemp, d_tbHContPadHash, d_tbHPosData, d_tbHSizePadHash, d_tbHIdxSizePadHash, d_tbHData, N, B, c, M, d_posicao);

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch wmkernel kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//=======fim executa o kernel=====================================================

	cout << "\n";

	//=======copia saidas do device p/ host=====================================================
	err = hipMemcpy(posicao, d_posicao, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector posicao from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	/*
	err = hipMemcpy(teste, d_teste, sizeChar, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector d_teste from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	*/



	for (int i = 0; i < N; i++) {
		//if(posicao[i]>0) 
		printf("posicao[%d]  %d ha um match \n", i, posicao[i]);

	}
	//=======fim copia saidas do device p/ host=====================================================

	hipFree(d_dataTemp);
	hipFree(d_tbHContPadHash);
	hipFree(d_tbHPosData);
	hipFree(d_tbHSizePadHash);
	hipFree(d_tbHIdxSizePadHash);
	hipFree(d_tbHData);
	hipFree(d_posicao);

	std::cout << std::endl;
	std::cout << "Fim exec" << std::endl;

}


void execCPU() {

	std::string dataTemp = "ABEDEDAGB";

	//int N = conttbHData;
	int N = dataTemp.size();

	int pos = 0;

	while (pos <= N - B) {
		//posicao[index] = index;

		int hash = 0;
		for (int i = pos; i < pos+B; i++) {
			hash = hash + dataTemp[i] * (c ^ (i - pos));
			hash = hash % M;
		}
		//posicao[index] = hash;

		int tamArrPadrao = 0;
		for (unsigned t = 0; t < tbHContPadHash[hash]; t++) {
			int match = 1;

			int posIniPadrao = tbHPosData[hash] - 1;
			tamArrPadrao = tamArrPadrao + tbHSizePadHash[tbHIdxSizePadHash[hash] + t];
			int tamPadrao = tbHSizePadHash[tbHIdxSizePadHash[hash] + t];
			int _final = (pos + B) - 1;
			int _final2 = tamPadrao;
			for (int ch = tamArrPadrao; ch >= 0; --ch) {
				if (_final2 - 1 >= 0) {

					if (tbHData[posIniPadrao + ch - 1] == dataTemp[_final]) {
						_final = _final - 1;
						_final2 = _final2 - 1;

					}
					else {
						match = 0;
						break;
					}
				}
				else {

					break;
				}
			}
			if (match) {
				//posicao[index] = _final + 1;
				cout << "Ha um match a partir da posicao " << _final + 1 << " do  pacote capturado" << endl;


			}
		}

		pos = pos + 1;
	}
}

int
main(void)
{
	THREADS_PER_BLOCk = 32;

	std::string dir = "C:/Users/danie/Documents/Projects/TCC/GPUNIDS_AC_WM/rulesDebug.txt";
	std::string dir2 = "C:/Users/danie/Documents/Projects/TCC/Wu-Wanber/teste/padroes/padroes.txt";

	std::ifstream file(dir);
	std::string str;
	while (std::getline(file, str)) {
		//std::cout << str << "n";
		padroes.push_back(str);
	}


	m = TmhMenorPadrao();
	nP = padroes.size();
	//cout << "nP: " << nP << "\n";
	M = TmhTodosPadroes();
	B = BlcStr();

	for (int i = 0; i < nP; i++) {

		tbHDataSize = tbHDataSize + padroes.at(i).size() * (padroes.at(i).size() - (B - 1));

	}

	arrHashEsgotados = std::vector<int> (tbHDataSize);

	size_t sizeChar = tbHDataSize * sizeof(char);
	size_t sizeInt = M * sizeof(int);
	//tbHData = (char*)calloc(tbHDataSize, sizeChar);


	//================aloca host tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData
	tbHPosData = (int*)malloc(sizeInt);
	//tbHContPadHash = (int*) malloc(sizeInt);
	tbHContPadHash = (int*)calloc(M, sizeInt);
	tbHSizePadHash = (int*)malloc(sizeInt);
	tbHIdxSizePadHash = (int*)malloc(sizeInt);
	//tbHData = (char*) malloc(sizeChar);
	tbHData = (char*)calloc(tbHDataSize, sizeof(char)); //tbHData = (char*)calloc(tbHDataSize, sizeChar);
	//================fim aloca host tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData

	
	vecPermu1_1();

	execCPU();


	size_t sizeChartb = tbHDataSize * sizeof(char);
	size_t sizeInttb = M * sizeof(int);

	hipError_t err = hipSuccess;
	//=========== aloca device d_tbHData/d_tbHContPadHash/d_tbHPosData/d_tbHSizePadHash/d_tbHIdxSizePadHash
	err = hipMalloc((void**)& d_tbHData, sizeChartb);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_tbHData (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void**)& d_tbHContPadHash, sizeInttb);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_tbHContPadHash (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void**)& d_tbHPosData, sizeInttb);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_tbHPosData (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void**)& d_tbHSizePadHash, sizeInttb);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_tbHSizePadHash (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void**)& d_tbHIdxSizePadHash, sizeInttb);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_tbHIdxSizePadHash (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//===========fim aloca device d_tbHData/d_tbHContPadHash/d_tbHPosData/d_tbHSizePadHash/d_tbHIdxSizePadHash

	//========== copia host->device tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData====
	err = hipMemcpy(d_tbHData, tbHData, sizeChartb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector tbHData from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_tbHContPadHash, tbHContPadHash, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector tbHContPadHash from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_tbHPosData, tbHPosData, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector tbHPosData from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_tbHSizePadHash, tbHSizePadHash, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector tbHSizePadHash from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_tbHIdxSizePadHash, tbHIdxSizePadHash, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector tbHIdxSizePadHash from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	//==========fim copia host->device tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData====
	
	

	return 0;

}

int TmhMenorPadrao() {

	std::vector<int> sizePadroes;
	for (int i = 0; i < padroes.size(); ++i) {
		sizePadroes.push_back(padroes[i].size());
	}
	auto minElem = std::min_element(std::begin(sizePadroes), std::end(sizePadroes));
	//std::cout<<minElem.operator*()+"\n"<<std::endl;


	return minElem.operator*();
}

int TmhTodosPadroes() {

	int TodosP = m * nP;

	return  TodosP;
}

double BlcStr() {

	double blcStr = log10(2 * M) / log10(c);
	//double B = std::ceil(blcStr);
	double Blc = std::round(blcStr);
	return Blc;
}


void vecPermu1_1() {

	
	char* dataTemp = NULL;
	//int sizeCharDT = B * sizeof(char);
	size_t sizeChar = B * sizeof(char);//size_t sizeChar = tbHDataSize * sizeof(char);
	//dataTemp = (char*) malloc(sizeCharDT);
	//dataTemp = (char*) calloc(B,sizeCharDT);
	dataTemp = (char*)calloc(B, sizeChar);//dataTemp = (char*)calloc(tbHDataSize, sizeChar);
	//vecpermu = std::vector<std::vector<std::string>>(nP,std::vector<std::string>());


	//tabelaHASH_ = std::vector<std::vector<std::string>> (M,std::vector<std::string>()); //2*nP

	for (int i = 0; i < padroes.size(); ++i) {
		std::string strTemp = padroes[i];
		std::string subStrPerm;

		int idxInicio = 0;
		int idxFim = B - 1;

		//cout<<"idxInicio: "<<idxInicio<<"\n";
		//cout<<"idxFim: "<<idxFim<<"\n";

		//std::cout << "padrao: " << strTemp << std::endl;

		while (idxFim < strTemp.length()) {

			subStrPerm = strTemp.substr(idxInicio, B);
			//cout<<"subStrPerm: "<<subStrPerm<<std::endl;
			int contDataTemp = 0;
			for (int p = 0; p < subStrPerm.size(); ++p) {
				//if(subStrPerm[p] != '\0'){
				dataTemp[p] = subStrPerm[p];
				contDataTemp++;
				//cout << dataTemp[p];
				//}
			}
			//std::cout << "erro" << std::endl;

			int hash = 0;
			
			for (int i = 0; i < contDataTemp; i++) {//for (int i = 0; i < strlen(dataTemp); i++) {
				hash = hash + dataTemp[i] * (c ^ (i));
				hash = hash % M;
			}

			//cout<<"hash: "<<hash<<std::endl;

			if (!(hashEsgotado(hash))) {
				PrecarregaTabelaHash(strTemp, hash, dataTemp);
			}



			//PrecarregaTabelaHash(strTemp, hash,dataTemp);
			//PrecarregaTabelaHash(subStrPerm,i);

			if (idxInicio == idxFim) {
				idxFim = idxFim + 1;
				idxInicio = idxFim;
			}
			else {
				idxInicio = idxInicio + 1;
				idxFim = idxFim + 1;
			}

		}

	}
	//std::cout <<"	padroes.size: "<<padroes.size()<<std::endl;
	//std::cout <<"	tabelaHASH_.size: "<<tabelaHASH_.size()<<std::endl;

	//cout << "fim vecPermu1_1" << endl;

	delete[] dataTemp;
	//free(dataTemp);

	std::cout << std::endl;
	std::cout << "Fim inicializacao de tabelas" << std::endl;
}


bool hashEsgotado(int hash) {


	if (std::find(std::begin(arrHashEsgotados), std::end(arrHashEsgotados), hash) != std::end(arrHashEsgotados)) {
		return true;
	}
	else {
		return false;
	}

}


void PrecarregaTabelaHash(std::string strTemp, int hash, char* dataTemp) {

	
	char* dataTemp2 = NULL;//char* dataTemp2;
	//int sizeCharDT2 = B * sizeof(char);
	size_t sizeChar = B * sizeof(char);//size_t sizeChar = tbHDataSize * sizeof(char);
	//dataTemp2 = (char*) malloc(sizeCharDT2);
	//dataTemp2 = (char*) calloc(B,sizeCharDT2);
	dataTemp2 = (char*)calloc(B, sizeChar);


	for (int j = 0; j < padroes.size(); j++) {
		bool hashEsgotadoPadraoAtual = false;
		std::string strTemp2 = padroes[j];



		int idxInicio2 = 0;
		int idxFim2 = B - 1;


		while (idxFim2 < strTemp2.length()) {
			std::string subStrPerm2 = strTemp2.substr(idxInicio2, B);


			int contdataTemp2 = 0;
			for (int p = 0; p < subStrPerm2.size(); ++p) {
				dataTemp2[p] = subStrPerm2[p];
				//cout<<dataTemp2[p];
				contdataTemp2++;
			}



			int hash2 = 0;
			for (int i = 0; i < contdataTemp2; i++) {//for (int i = 0; i < strlen(dataTemp); i++) {
				hash2 = hash2 + dataTemp2[i] * (c ^ (i));
				hash2 = hash2 % M;
			}


			/*
			if (indexDebug == 163009) {
				cout << endl;
				cout << "indexDebug: 163009" << endl;
				cout << "hash:"<<hash <<" - hash2: "<< hash2 << endl;
				cout << "strTemp:" << strTemp << " - strTemp2: " << strTemp2 << endl;
			}*/

			if (hash == hash2) {//if(hash == hash2 && !(hashEsgotado(hash))  ) {
				//cout << "hash==hash2" << endl;
				if (!hashEsgotadoPadraoAtual) {

					/*
					if (indexDebug == 163009) {
						cout << "indexDebug: 163009" << endl;
						cout << "hash nao esgotado" << endl;
					}*/
					
					if (!strTemp.compare(strTemp2)) {

						/*
						if (indexDebug == 163009) {
							cout << "indexDebug: 163009" << endl;
							cout << "strTemp==strTemp2" << endl;
						}*/
						

						int posIn = 0;
						int min = conttbHData; //int min = strlen(tbHData);

						//cout << " strlen(tbHData): " << min << endl;

						posIn = min;
						int max = strTemp.size() + conttbHData;//int max = strTemp.size() + strlen(tbHData);
						int idx = 0;
						//cout << "\n";



						//cout<<"min: "<<min<<"\n";
						//cout<<"max: "<<max<<"\n";
						for (int d = min; d < max; d++) {
							//if( strTemp[idx] != '\0'){
							tbHData[d] = strTemp[idx];
							idx = idx + 1;
							cout << "[" << d << "]: " << tbHData[d] << '\n';
							indexDebug = d;
							
							conttbHData = conttbHData + 1;
							//}
						}
						//cout<<">>>>>hash"<<hash<<"\n";
						//cout<<">>>>>crash1"<<"\n";

						if (tbHContPadHash[hash] == 0) {//if (tbHPosData[hash] <= 0) { //exe:  -3453456 (lixo de memoria)
							tbHPosData[hash] = posIn + 1;
							cout << "tbHPosData[" << hash << "]: " << tbHPosData[hash] << "  ";
						}

						int cont = tbHContPadHash[hash];
						tbHContPadHash[hash] = cont + 1;
						cout << "tbHContPadHash[" << hash << "]: " << tbHContPadHash[hash] << "  ";

						tbHSizePadHash[countSizePadHash] = strTemp.size();
						cout << "tbHSizePadHash[" << countSizePadHash << "]: " << tbHSizePadHash[countSizePadHash]							<< "  ";


						tbHIdxSizePadHash[hash] = countSizePadHash;
						cout << "tbHIdxSizePadHash[" << hash << "]: " << tbHIdxSizePadHash[hash] << "\n";

						hashEsgotadoPadraoAtual = true;
						countSizePadHash++;
					}else {

						/*
						if (indexDebug == 163009) {
							cout << "indexDebug: 163009" << endl;
							cout << "strTemp!=strTemp2" << endl;
						}*/
						

						int posIn = 0;
						int min = conttbHData; //int min = strlen(tbHData);

						posIn = min;
						int max = strTemp2.size() + conttbHData; //int max = strTemp2.size() + strlen(tbHData);
						int idx = 0;
						//cout << "\n";

						//cout<<"min: "<<min<<"\n";
						//cout<<"max: "<<max<<"\n";
						for (int d = min; d < max; d++) {
							//if(strTemp2[idx] != '\0'){
							tbHData[d] = strTemp2[idx];
							idx = idx + 1;
							cout << "[" << d << "]: " << tbHData[d] << '\n';
							conttbHData = conttbHData + 1;
							indexDebug = d;
							//}
						}
						//cout<<">>>>>hash"<<hash<<"\n";
						//cout<<">>>>>crash2"<<"\n";

						if (tbHContPadHash[hash] == 0) {//if (tbHPosData[hash] <= 0) { //exe:  -3453456 (lixo de memoria)
							tbHPosData[hash] = posIn + 1;
							cout << "tbHPosData[" << hash << "]: " << tbHPosData[hash] << "  ";
						}


						int cont = tbHContPadHash[hash];
						tbHContPadHash[hash] = cont + 1;
						cout << "tbHContPadHash[" << hash << "]: " << tbHContPadHash[hash] << "  ";


						tbHSizePadHash[countSizePadHash] = strTemp2.size();
						cout << "tbHSizePadHash[" << countSizePadHash << "]: " << tbHSizePadHash[countSizePadHash]							<< "  ";


						//if(tbHIdxSizePadHash[hash] < 0) {
						//    tbHIdxSizePadHash[hash] = countSizePadHash;
						//    cout << "tbHIdxSizePadHash[" << hash << "]: " << tbHIdxSizePadHash[hash] << "\n";
						//}

						hashEsgotadoPadraoAtual = true;
						countSizePadHash++;
					}
				}
				//cout << "hash esgotado" << endl;
			}
			//cout << "hash!=hash2" << endl;


			if (idxInicio2 == idxFim2) {
				idxFim2 = idxFim2 + 1;
				idxInicio2 = idxFim2;
			}
			else {
				idxInicio2 = idxInicio2 + 1;
				idxFim2 = idxFim2 + 1;
			}

		}


		if (j == padroes.size() - 1) {
			arrHashEsgotados.push_back(hash);
		}


	}

	//cout << "fim vecPermu1_1" << endl;
	delete[] dataTemp2;
	//free(dataTemp2);
}



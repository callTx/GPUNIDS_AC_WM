#include "hip/hip_runtime.h"
#include"WMGPUinspection.h"


WMGPUInspection::WMGPUInspection()
{

}

WMGPUInspection::~WMGPUInspection()
{
	free(tbHData);
}

void WMGPUInspection::cuda_sample_random_ints(int* x,int N){

	for(int i=0; i<N;i++){
		x[i]=rand();
	}
}

void WMGPUInspection::cuda_sample_print_ints(int* x,int N){

	for(int i=0; i<N;i++){
		std::cout<<"x["<<i<<"]: "<<x[i]<<std::endl;
	}
}

/*
 __global__ void add(int *x,int *y,int *z){
	*z= *x+ *y; //Heterogeneous computing
	//c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; //Blocks
	//c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x]; //Threads
	
	//int index = threadIdx.x + blockIdx.x * blockDim.x; //
	//if(index < n) c[index] = a[index] + b[index];
	
		//c [ 0 ] = a[0] + b[0]  ----- c[1] = a[1] +b[1]
	
} */



__global__ void
wmkernel(char *dataTemp,int *tbHContPadHash,int *tbHPosData,int *tbHSizePadHash,int *tbHIdxSizePadHash,char *tbHData,int N,double B,int c, int M, int *posicao){	

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	
		
	
	
	if(index < N-B+1){
		//posicao[index] = index;
		
		int hash=0;
		for(int i=index;i<index+B;i++){
			hash = hash + dataTemp[i]*(c^(i-index));
			hash = hash % M;
		}
		//posicao[index] = hash;

		int tamArrPadrao = 0;
		for(unsigned t =0; t< tbHContPadHash[hash]; t++){
			int match = 1;

			int posIniPadrao = tbHPosData[hash]-1;
			tamArrPadrao = tamArrPadrao+ tbHSizePadHash[ tbHIdxSizePadHash[hash] +t ];
			int tamPadrao = tbHSizePadHash[ tbHIdxSizePadHash[hash] +t ];
			int _final = (index + B) -1; 
			int _final2 = tamPadrao;
			for (int ch = tamArrPadrao; ch>=0; --ch){
				if(_final2 -1 >=0){

					if(tbHData[posIniPadrao+ch-1] == dataTemp[_final]){
						_final = _final - 1;
						_final2 = _final2 -1;

					}else{
						match = 0;
						break;
					}
				}else{
					
					break;
				}
			}
			if (match){
				posicao[index] = _final +1;
			

			}
		}
		

	}
	
	
}





void WMGPUInspection::exec(std::shared_ptr<Packet> pkt){
    
	#ifdef VERBOSE
			struct timeval start, end, diff;
			gettimeofday(&end, nullptr);
			timersub(&end, &(pkt.get()->virtualTime), &diff);
			this->lock();
			pkt->computeStatistics(this->getStats());
			this->getStats()->sumWaitingTime += diff.tv_sec * 1000.0 + diff.tv_usec / 1000.0;
			this->unlock();

			if( gettimeofday(&start, nullptr) != 0)
			{
				std::cerr << "Fail to get current time" << std::endl;
				exit(-1);
			}
	#endif

	pkt.get()->init();
	if(nextInput && pkt.get()->size_payload_ > 0){
			nextInput = false;
			
			hipError_t err = hipSuccess;

			//========== aloca host, aloca device dataTemp ==========
			char* dataTemp;
			dataTemp = reinterpret_cast<char*>(pkt.get()->payload_);;
			
			int N = strlen(dataTemp);
			size_t sizeChar = N * sizeof(char);

			char* d_dataTemp = NULL;
			err = hipMalloc((void **)&d_dataTemp, sizeChar);
			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to allocate device vector d_dataTemp (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
			//==========fim aloca host, aloca device dataTemp ==========


			//============ copia device dataTemp ===============================
			err = hipMemcpy(d_dataTemp, dataTemp, sizeChar, hipMemcpyHostToDevice);

			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to copy vector dataTemp from host to device (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
			//============fim copia device dataTemp ===============================



			//=========== aloca host,aloca device posicao===============
			size_t size = N * sizeof(int);
			int *posicao;
			int *d_posicao = NULL;

			posicao = (int *)calloc(N,size);
			hipMalloc((void **)&d_posicao, size);
			//===========fim: aloca host,aloca device posicao===============


			//======aloca host,aloca device teste===========
			/*
			char *teste;
			char *d_teste = NULL;

			teste = (char *)calloc(N,sizeChar);
			hipMalloc((void **)&d_teste, sizeChar);
			*/
			//======fim aloca host,aloca device teste===========



			//======== verifica se N e multiplo de THREADS_PER_BLOCk ======================
			NUMBER_BLOCKS = N/ THREADS_PER_BLOCk;
			cout<<"\n";
			cout<<"NUMBER_BLOCKS: "<<NUMBER_BLOCKS<<"\n";

			int X_;
			int Y_;
			if(N%THREADS_PER_BLOCk ==0){
				cout<<"N/ THREADS_PER_BLOCk: "<<N/ THREADS_PER_BLOCk<<" e multiplo";
				X_ = N/THREADS_PER_BLOCk;
				Y_ = THREADS_PER_BLOCk;
			}else{
				cout<<"N/ THREADS_PER_BLOCk: "<<N/ THREADS_PER_BLOCk<<" noa e multiplo";
				X_ =(NUMBER_BLOCKS + THREADS_PER_BLOCk -1)/THREADS_PER_BLOCk;
				Y_ = THREADS_PER_BLOCk;
			}
			//========fim verifica se N e multiplo de THREADS_PER_BLOCk ======================


			//=======executa o kernel=====================================================
			wmkernel<<< X_, Y_  >>>(d_dataTemp,d_tbHContPadHash,d_tbHPosData,d_tbHSizePadHash,d_tbHIdxSizePadHash,d_tbHData,N, B, c, M, d_posicao);	

			err = hipGetLastError();

			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to launch wmkernel kernel (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
			//=======fim executa o kernel=====================================================

			cout<<"\n";


			//=======copia saidas do device p/ host=====================================================
			err = hipMemcpy(posicao, d_posicao, size, hipMemcpyDeviceToHost);
			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to copy vector posicao from device to host (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			/*
			err = hipMemcpy(teste, d_teste, sizeChar, hipMemcpyDeviceToHost);
			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to copy vector d_teste from device to host (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
			*/

			

			for(int i=0;i<N;i++){
				//if(posicao[i]>0) 
				printf("posicao[%d]  %d ha um match \n",i,posicao[i]);
				
			}
			//=======fim copia saidas do device p/ host=====================================================

			hipFree(d_dataTemp);
			hipFree(d_tbHContPadHash);
			hipFree(d_tbHPosData);
			hipFree(d_tbHSizePadHash);
			hipFree(d_tbHIdxSizePadHash);
			hipFree(d_tbHData);
			hipFree(d_posicao);
		



			#ifdef VERBOSE
				//:::::::::::::      
				//:::::::::::::      blocks number = N / THREADS_PER_BLOCK  

				//add<<<(N+M-1)/M, M >>>(d_a,d_b,d_c,N); //threads_per_block with vector which is not multiple of the blockDim.x = M
				//nextInput = false;
				//std::cout<<">>>>>>>>>>>>>>>>payload_[ ]: " <<reinterpret_cast<char*>( pkt.get()->payload_) <<std::endl;
				//std::cout<< "size: " << pkt.get()->size_payload_ <<std::endl;
				//this->excWuManberCore( reinterpret_cast<char*>(pkt.get()->payload_) );
				//cout<<"dataTemp: "<<dataTemp<<"\n";
				//int pos = m;
				//int pref = m - B2;
				//cout<< ">>>>>>>>>>>WuManberCore1_1: " <<endl;	
				//std::string dataTemp = nextData;
				//std::string dataTemp = "ABEDEDABG";
				//std::cout<<"dataTemp: "<<dataTemp <<std::endl;
				//int N = dataTemp.length();
				//cout <<">>>>>>>>>>>>>N: "<<N<< endl;
				//cout <<">>>>>>>>>>>>>pos: "<<pos<< endl;
				//cout <<">>>>>>>>>>>>>pref: "<<pref<< endl;
			#endif
		
	}

	#ifdef VERBOSE

			if( gettimeofday(&end, nullptr) != 0)
			{
				std::cerr << "Fail to get current time" << std::endl;
				exit(-1);
			}
			timersub(&end, &start, &diff);
			this->lock();
			this->getStats()->sumProcTime += diff.tv_sec * 1000.0 + diff.tv_usec / 1000.0;
			this->unlock();
	#endif
	
}


void WMGPUInspection::excWuManberCore(std::string  nextData){	
}

int WMGPUInspection::buildMatchingMachine(std::string rules, int bufferSize, int nThreads)
{
		//std::cout<<"nThreads: "<<nThreads<<std::endl;
		
		// (14-15/05) getilne -- TmhMenorPadrao() -- padroes.size() -- TmhTodosPadroes() -- BlcStr -- B2=B --  tabeleIdxHash() -- vecPermu() -- TabelaHash() 

		//TODO (20/05) buildMatchingMachine

		std::string line;
    	ifstream file(rules.c_str(), std::ifstream::in);
    	maxs_ = 0;
    	while(getline(file,line))
    	{
        	padroes.push_back(line);    
        	maxs_ += line.size();
		}
		
		m = TmhMenorPadrao();
		nP = padroes.size();
		M = TmhTodosPadroes();
		B = BlcStr(); 
		B2 = B;
		#ifdef VERBOSE
			cout<< "Inicializando tabelas........................... " <<endl;	
		#endif

		tabelaIdxHash();
		//tabelaShift();

		#ifdef VERBOSE
			cout<< "Tabelas inicializadas........................... " <<endl;	
		#endif

		return 1;

		/*
		int *a,*b,*c;
		//int a,b,c;
		int *d_a,*d_b,*d_c; 
		// size= sizeof(int);
		//std::cout<<"N: "<<N<<std::endl;
		int size = N * sizeof(int);

		//Aloca espaco para o device copiar de a,b,c
		hipMalloc( (void **)&d_a,size );		
		hipMalloc( (void **)&d_b,size );		
		hipMalloc( (void **)&d_c,size );
		
		a = (int*)malloc(size);
		cuda_sample_random_ints(a,N);
		b= (int*)malloc(size);
		cuda_sample_random_ints(b,N);
		c= (int*)malloc(size);


		//a =2;
		//b =7;
		//copia a,b para o device
		hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);		
		hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);	

		//Launch o kernel add() na GPU
		//add<<<N,1>>>(d_a,d_b,d_c); //blocks
		//add<<<1,N>>>(d_a,d_b,d_c); //threads

		
		//:::::::::::::      N/THREADS_PER_BLOCK  = blocks number
		
		//add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c); //threads_per_block with vector multiple of blockDim.x = M 

		add<<<(N+M-1)/M, M >>>(d_a,d_b,d_c,N); //threads_per_block with vector which is not multiple of the blockDim.x = M	

		//Copiar os resultados de volta p/ host
		hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

		//cuda_sample_print_ints(c,N);
		//std::cout<<"c: "<<c<<std::endl;
		
		//Cleanup
		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

		//char *hipGetErrorString(hipError_t);
		//printf("%s\n",hipGetErrorString(hipGetLastError()));

		return 0;
		*/
		
}

int WMGPUInspection::buildMatchingMachine1_1(std::string rules,int bufferSize, int nThreads){
	THREADS_PER_BLOCk = nThreads;
	
	std::string line;
	ifstream file(rules.c_str(), std::ifstream::in);
	maxs_ = 0;
	

	while(getline(file,line))
	{
		padroes.push_back(line);
		
		maxs_ += line.size();
	}

	

	m = TmhMenorPadrao();
	
	nP = padroes.size();

	cout<<"nP: "<<nP<<"\n";


	M = TmhTodosPadroes();

	B = BlcStr(); 


	B2 = B;

	tbHDataSize = 0;
	cout<<"B: "<<B<<"\n";
	
	for (int i =0;i< nP;i++) {
		
        tbHDataSize = tbHDataSize + padroes.at(i).size()*(padroes.at(i).size() - (B-1));
	}
	
	//cout<<"tbHDataSize: "<<tbHDataSize<<"\n";

	#ifdef VERBOSE
		cout<< "Inicializando tabelas........................... " <<endl;	
	#endif


	size_t sizeChar = tbHDataSize * sizeof(char);	
	size_t sizeInt = M * sizeof(int);
	
	
	//================aloca host tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData
    tbHPosData = (int*) malloc(sizeInt);
	//tbHContPadHash = (int*) malloc(sizeInt);
	tbHContPadHash = (int*) calloc(M,sizeInt);
    tbHSizePadHash = (int*) malloc(sizeInt);
    tbHIdxSizePadHash = (int*) malloc(sizeInt);
	//tbHData = (char*) malloc(sizeChar);
	tbHData = (char*) calloc(tbHDataSize,sizeChar);
	//================fim aloca host tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData
	
	
	vecPermu1_1();
	
	size_t sizeChartb = tbHDataSize * sizeof(char);	
	size_t sizeInttb = M * sizeof(int);
	

	hipError_t err = hipSuccess;
	//=========== aloca device d_tbHData/d_tbHContPadHash/d_tbHPosData/d_tbHSizePadHash/d_tbHIdxSizePadHash
	err = hipMalloc((void **)&d_tbHData, sizeChartb);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_tbHData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMalloc((void **)&d_tbHContPadHash, sizeInttb);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_tbHContPadHash (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMalloc((void **)&d_tbHPosData, sizeInttb);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_tbHPosData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMalloc((void **)&d_tbHSizePadHash, sizeInttb);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_tbHSizePadHash (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMalloc((void **)&d_tbHIdxSizePadHash, sizeInttb);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_tbHIdxSizePadHash (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	//===========fim aloca device d_tbHData/d_tbHContPadHash/d_tbHPosData/d_tbHSizePadHash/d_tbHIdxSizePadHash

		
	//========== copia host->device tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData====
	err = hipMemcpy(d_tbHData, tbHData, sizeChartb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector tbHData from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpy(d_tbHContPadHash, tbHContPadHash, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector tbHContPadHash from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpy(d_tbHPosData, tbHPosData, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector tbHPosData from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpy(d_tbHSizePadHash, tbHSizePadHash, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector tbHSizePadHash from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpy(d_tbHIdxSizePadHash, tbHIdxSizePadHash, sizeInttb, hipMemcpyHostToDevice);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector tbHIdxSizePadHash from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	//==========fim copia host->device tbHPosData/tbHContPadHash/tbHSizePadHash/tbHIdxSizePadHash/tbHData====

	#ifdef VERBOSE
		/*tbHData = "\000";
		
		cout<<"strlen(tbHData): "<<strlen(tbHData)<<"\n";
		for (int i =0;i< strlen(tbHData) ;i++) {
			cout<<"tbHData["<<i<<"]"<< tbHData[i]<<"\n";
		}
		*/
		/*cout<<">>>>>>>padroes: "<<endl;
		for(int i=0 ; i<padroes.size() ; i++){

			cout<<">>>>>>>>>>>>>>>" <<padroes[i]<<endl;
		}*/
		//cout<<">>>>>>>>>>> m: "<< m <<endl; 
		//cout<<">>>>>>>>>>> nP: "<< nP <<endl; 
		//cout<<">>>>>>>>>>> M: "<< M <<endl; 
		//cout<<">>>>>>>>>>> B: "<< B <<endl; 
		//tabelaIdxHash();
		//cout<< "crash malloc"<<"\n";
		//cout<< "crash vecPermu"<<"\n";
		//TabelaHash();	
		//cout<< ">>>>>>>>>>>tabelaidx.size(): "<<tabelaidx.size() <<endl;	
		//tabelaShift();
		//cout<< ">>>>>>>>>>>tabelaSHIFT: "<<tbShift.size() <<endl;	
		cout<< "Tabelas inicializadas........................... " <<endl;	
	#endif
	 
	return 1;

}

int WMGPUInspection::TmhMenorPadrao(){
	
    std::vector<int> sizePadroes;
    for (int i = 0; i <padroes.size() ; ++i) {
        sizePadroes.push_back(padroes[i].size());
    }
    auto minElem = std::min_element(std::begin(sizePadroes), std::end(sizePadroes));
    //std::cout<<minElem.operator*()+"\n"<<std::endl;

   
    return minElem.operator*();
}

int WMGPUInspection::TmhTodosPadroes(){

    int TodosP = m * nP;

    return  TodosP;
}

int WMGPUInspection::BlcStr(){

	double blcStr = log10(2*M)/log10(c);
	//double B = std::ceil(blcStr);
	double Blc = std::round(blcStr);
	return Blc;
}

void WMGPUInspection::tabelaIdxHash(){

	vecPermu();
	
	TabelaHash();


}

void WMGPUInspection::PrecarregaTabelaHash(std::string strTemp, int hash,char* dataTemp){
	
	
	char* dataTemp2;
	//int sizeCharDT2 = B * sizeof(char);
	size_t sizeChar = tbHDataSize * sizeof(char);
	//dataTemp2 = (char*) malloc(sizeCharDT2);
	//dataTemp2 = (char*) calloc(B,sizeCharDT2);
	dataTemp2 = (char*) calloc(tbHDataSize,sizeChar);
	

	for(int j=0;j<padroes.size();j++) {
        bool hashEsgotadoPadraoAtual = false;
		std::string strTemp2 = padroes[j];

		

		int idxInicio2 = 0;
		int idxFim2 = B - 1;

		
		while(idxFim2<strTemp2.length()) {
			std::string subStrPerm2 = strTemp2.substr(idxInicio2,B);

			

			for (int p = 0; p <subStrPerm2.size(); ++p) {
                dataTemp2[p] = subStrPerm2[p];
                //cout<<dataTemp2[p];
			}

			

			int hash2 = 0;
            for (int i = 0; i < strlen(dataTemp); i++) {
                hash2 = hash2 + dataTemp2[i] * (c ^ (i));
                hash2 = hash2 % M;
			}
			
			

			if(hash == hash2   ) {//if(hash == hash2 && !(hashEsgotado(hash))  ) {

                if(!hashEsgotadoPadraoAtual) {
                    if (!strTemp.compare(strTemp2)) {

							

							int posIn = 0;
							int min = strlen(tbHData);
							
							cout<<" strlen(tbHData): "<<min<<endl;
							
                            posIn = min;
                            int max = strTemp.size() + strlen(tbHData);
                            int idx = 0;
							cout << "\n";

							
							
							//cout<<"min: "<<min<<"\n";
							//cout<<"max: "<<max<<"\n";
							for (int d = min; d < max; d++) {
								//if( strTemp[idx] != '\0'){
									tbHData[d] = strTemp[idx];
									idx = idx + 1;
									cout << "[" << d << "]: " << tbHData[d] << '\n';
								//}
							}
							//cout<<">>>>>hash"<<hash<<"\n";
							//cout<<">>>>>crash1"<<"\n";

							if (tbHContPadHash[hash] == 0) {//if (tbHPosData[hash] <= 0) { //exe:  -3453456 (lixo de memoria)
                                tbHPosData[hash] = posIn + 1;
                                cout << "tbHPosData[" << hash << "]: " << tbHPosData[hash] << "  ";
							}
							
							int cont = tbHContPadHash[hash];
                            tbHContPadHash[hash] = cont + 1;
							cout << "tbHContPadHash[" << hash << "]: " << tbHContPadHash[hash] << "  ";
							
							tbHSizePadHash[countSizePadHash] = strTemp.size();
                            cout << "tbHSizePadHash[" << countSizePadHash << "]: " << tbHSizePadHash[countSizePadHash]
                                 << "  ";


                            tbHIdxSizePadHash[hash] = countSizePadHash;
                            cout << "tbHIdxSizePadHash[" << hash << "]: " << tbHIdxSizePadHash[hash] << "\n";

                            hashEsgotadoPadraoAtual = true;
                            countSizePadHash++;
					}else{
						int posIn = 0;
                        int min = strlen(tbHData);

                        posIn = min;
                        int max = strTemp2.size() + strlen(tbHData);
                        int idx = 0;
                        cout << "\n";
						
						//cout<<"min: "<<min<<"\n";
						//cout<<"max: "<<max<<"\n";
                        for (int d = min; d < max; d++) {
							//if(strTemp2[idx] != '\0'){
								tbHData[d] = strTemp2[idx];
								idx = idx + 1;
								cout << "[" << d << "]: " << tbHData[d] << '\n';
							//}
                        }
						//cout<<">>>>>hash"<<hash<<"\n";
						//cout<<">>>>>crash2"<<"\n";

                        if (tbHContPadHash[hash] == 0) {//if (tbHPosData[hash] <= 0) { //exe:  -3453456 (lixo de memoria)
                            tbHPosData[hash] = posIn + 1;
                            cout << "tbHPosData[" << hash << "]: " << tbHPosData[hash] << "  ";
                        }


                        int cont = tbHContPadHash[hash];
                        tbHContPadHash[hash] = cont + 1;
                        cout << "tbHContPadHash[" << hash << "]: " << tbHContPadHash[hash] << "  ";


                        tbHSizePadHash[countSizePadHash] = strTemp2.size();
                        cout << "tbHSizePadHash[" << countSizePadHash << "]: " << tbHSizePadHash[countSizePadHash]
                             << "  ";

                        
                        //if(tbHIdxSizePadHash[hash] < 0) {
                        //    tbHIdxSizePadHash[hash] = countSizePadHash;
                        //    cout << "tbHIdxSizePadHash[" << hash << "]: " << tbHIdxSizePadHash[hash] << "\n";
                        //}

                        hashEsgotadoPadraoAtual = true;
                        countSizePadHash++;
					}
				}
			}
			

			if (idxInicio2==idxFim2){
                idxFim2 = idxFim2 + 1;
                idxInicio2 = idxFim2;
            }else{
                idxInicio2 = idxInicio2 + 1;
                idxFim2 = idxFim2 + 1;
            }

		}
		

		if(j == padroes.size()-1){
            arrHashEsgotados.push_back(hash);
        }

	
	}
	
	//delete[] dataTemp2;
	free(dataTemp2);
}

bool WMGPUInspection::hashEsgotado(int hash){


    if (std::find(std::begin(arrHashEsgotados), std::end(arrHashEsgotados), hash) != std::end(arrHashEsgotados)){
        return true;
    }else{
        return false;
    }

}

void WMGPUInspection::vecPermu1_1(){

	char* dataTemp;
	//int sizeCharDT = B * sizeof(char);
	size_t sizeChar = tbHDataSize * sizeof(char);
	//dataTemp = (char*) malloc(sizeCharDT);
	//dataTemp = (char*) calloc(B,sizeCharDT);
	dataTemp = (char*) calloc(tbHDataSize,sizeChar);
    //vecpermu = std::vector<std::vector<std::string>>(nP,std::vector<std::string>());

	
	//tabelaHASH_ = std::vector<std::vector<std::string>> (M,std::vector<std::string>()); //2*nP

    for (int i = 0; i < padroes.size() ; ++i) {
        std::string strTemp = padroes[i];
		std::string subStrPerm;
		
        int idxInicio = 0;
		int idxFim = B -1;
		
		//cout<<"idxInicio: "<<idxInicio<<"\n";
		//cout<<"idxFim: "<<idxFim<<"\n";

		std::cout <<"padrao: "<<strTemp<<std::endl;
		
        while(idxFim<strTemp.length()){
			
			subStrPerm = strTemp.substr(idxInicio,B);
			//cout<<"subStrPerm: "<<subStrPerm<<std::endl;
			for (int p = 0; p <subStrPerm.size(); ++p) {
				//if(subStrPerm[p] != '\0'){
                	dataTemp[p] = subStrPerm[p];
					cout<<dataTemp[p];
				//}
			}
			

			int hash = 0;
            for (int i = 0; i < strlen(dataTemp); i++) {
                hash = hash + dataTemp[i] * (c ^ (i));
                hash = hash % M;
			}
			
			//cout<<"hash: "<<hash<<std::endl;
			
			if(!(hashEsgotado(hash))){
                PrecarregaTabelaHash(strTemp, hash,dataTemp);
			}
			
			
			
			//PrecarregaTabelaHash(strTemp, hash,dataTemp);
			//PrecarregaTabelaHash(subStrPerm,i);

            if (idxInicio==idxFim){
                idxFim = idxFim + 1;
                idxInicio = idxFim;
            }else{
                idxInicio = idxInicio + 1;
                idxFim = idxFim + 1;
            }

        }
		
    }
	//std::cout <<"	padroes.size: "<<padroes.size()<<std::endl;
	//std::cout <<"	tabelaHASH_.size: "<<tabelaHASH_.size()<<std::endl;

	//delete[] dataTemp;
	free(dataTemp);
}

void WMGPUInspection::vecPermu(){

    vecpermu = std::vector<std::vector<std::string>>(nP,std::vector<std::string>());
 
    for (int i = 0; i < padroes.size() ; ++i) {
        std::string strTemp = padroes[i];
        std::string subStrPerm;

        int idxInicio = 0;
        int idxFim = B -1;
        while(idxFim<strTemp.length()){
            

            subStrPerm = strTemp.substr(idxInicio,B);

            


            if (idxInicio==idxFim){
                
               vecpermu[i].push_back(subStrPerm);
                idxFim = idxFim + 1;
                idxInicio = idxFim;
            }else{
                
                vecpermu[i].push_back(subStrPerm);
               
                idxInicio = idxInicio + 1;
                idxFim = idxFim + 1;

            }

        }

    }


}

void WMGPUInspection::TabelaHash(){
	
	int vpSize = 0;
	for(int j= 0; j< vecpermu.size();j++){
		vpSize = vpSize + vecpermu[j].size();
 	}

 	tabelaidx = std::vector<std::vector<std::string>> (vpSize, std::vector<std::string>());

 	for (int k = 0; k < vecpermu.size(); ++k) {
        	int len  = vecpermu[k].size();
        	for (int i = 0; i < len ; ++i) {
            		int h  = Hash( vecpermu[k].at(i) ,c,M);
            		std::string str = vecpermu[k].at(i);
            		if(h+1<=tabelaidx.size()) {
                		if (std::find(tabelaidx[h].begin(),tabelaidx[h].end(),str) == tabelaidx[h].end() ){
                    			tabelaidx[h].push_back(str);
                		}

            		}else{
                		tabelaidx.resize(h + 1);

                		tabelaidx[h].push_back(str);

            		}

        	}
		
	}

	int sZ = tabelaidx.size()-1;
   	if (tabelaidx[sZ].empty()){
        	tabelaidx[sZ].push_back("*");
	}else{
		tabelaidx.resize(sZ+2);
		tabelaidx[sZ+1].push_back("*");
	}

}

int WMGPUInspection::Hash(std::string strBloco, int c, int M){
    
    int hash = 0;
    for (int i = 0; i <strBloco.length() ; ++i) {
        hash = hash + strBloco[i] * pow(c,i);
		hash = hash % M;
    }

    return hash;
}

void WMGPUInspection::tabelaShift(){
	
    tbShift =  std::vector<int>(tabelaidx.size(),int());
    
    for (int b1 = 0; b1 <tabelaidx.size() ; ++b1) {
        if (!tabelaidx[b1].empty()) {
            for (int b2 = 0; b2 < tabelaidx[b1].size(); ++b2) {
                std::string blocoStr = tabelaidx[b1].at(b2);
                std::vector<int> min;
               
                for (int p = 0; p < padroes.size(); ++p) {
                    

                    std::string pdr;
                    std::vector<char> writable(padroes.at(p).begin(), padroes.at(p).end());
                    writable.push_back('\0');
                    pdr = &*writable.begin();
                    std::string s;

                    int q=0;

                    
                    int idxInicio = 0;
                    int idxFim = B - 1;
                    while (idxFim < pdr.length()){
                        
                        s = pdr.substr(idxInicio,B);
                        if (!blocoStr.find(s)){
                            q = idxFim + 1;

                        }
                        if (idxInicio == idxFim){
                            idxFim = idxFim + 1;
                            idxInicio = idxFim;
                        }else{
                            idxInicio = idxInicio+1;
                            idxFim = idxFim + 1;
                        }
                        if (idxFim == pdr.length() && q > 0){
                            while ((m - q) < 0) {
                                q = q - 1;
                            }

                            min.push_back(m - q);
                        }
                    }
                    

                } 
                if (min.empty()){
                    tbShift.insert(tbShift.begin()+b1,m - B +1);
                    tbShift.erase(tbShift.begin()+b1+1);
		    /*
		    if(b1 == 10542){
		    	#ifdef VERBOSE
				//cout<< ">>>>>>>>>>>tbShift[10542]: "<<tbShift[10542] <<endl;	
	            	#endif
		    }
		    */
                }else {
                    std::vector<int>::iterator result = std::min_element(std::begin(min), std::end(min));
                    tbShift.insert(tbShift.begin() + b1, result.operator*());
                    tbShift.erase(tbShift.begin() + b1 + 1);
		   /* 
		    if(b1 == 10542){
		    	#ifdef VERBOSE
				//cout<< ">>>>>>>>>>>tbShift[10542]: "<<tbShift[10542] <<endl;	
	            	#endif
		    }
		  */
                }
                
            }
        }


    }


}

std::vector<int> WMGPUInspection::AdcTodosPadroesTabela(std::string str){

	std::vector<int> listaPadroes;
    	for (int k = 0; k <padroes.size() ; ++k) {
        	if (std::find(vecpermu[k].begin(),vecpermu[k].end(),str) != vecpermu[k].end() ){

            		listaPadroes.push_back(k);
        	}
    	}
	return listaPadroes;
}

void WMGPUInspection::MatchPadroes(std::vector<std::string> listPotenPadr, std::string subData){

    for (unsigned t = 0; t <listPotenPadr.size() ; ++t) { 
        bool match = true;
        std::string str =listPotenPadr.at(t); 
        int _final = subData.size()-1;
        //int _final = dataTemp.size()-1;
        for (int ch = listPotenPadr.at(t).length()-1; ch>=0 ; --ch) { 
		if(_final>=0){
            		if( str.at(ch) ==  subData.at(_final)) {
                		_final = _final - 1;
            		}else{
                		match = false;
                		break;
            		}
		}else{
			match= false;
			break;
		}		

        }

        if (match){
            //int posicao = _final+1;
    	    #ifdef VERBOSE
            		//std::cout<<"\r"<<">>>>>>>>>>>>  Na posicao "<<posicao<<", ha um match com o padrao: "<<listPotenPadr.at(t)<<std::endl;
    	    #endif
		
        }
    }
}
